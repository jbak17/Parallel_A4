#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "bmpfile.h"

/*Mandelbrot values*/
#define RESOLUTION 8700.0
#define XCENTER -0.55
#define YCENTER 0.4
#define MAX_ITER 1000/2

/*Colour Values*/
#define COLOUR_DEPTH 255
#define COLOUR_MAX 100.0
#define GRADIENT_COLOUR_MAX 200.0

#define FILENAME "my_mandelbrot_fractal3.bmp"
#define BLOCKSIZE 16

/**
  * Computes the color gradiant
  * color: the output vector
  * x: the gradiant (beetween 0 and 360)
  * min and max: variation of the RGB channels (Move3D 0 -> 1)
  * Check wiki for more details on the colour science: en.wikipedia.org/wiki/HSL_and_HSV
  */
void GroundColorMix(double* color, double x, double min, double max)
{
	/*
	 * Red = 0
	 * Green = 1
	 * Blue = 2
	 */
	double posSlope = (max-min)/60;
	double negSlope = (min-max)/60;

	if( x < 60 )
	{
		color[0] = max;
		color[1] = posSlope*x+min;
		color[2] = min;
		return;
	}
	else if ( x < 120 )
	{
		color[0] = negSlope*x+2.0*max+min;
		color[1] = max;
		color[2] = min;
		return;
	}
	else if ( x < 180  )
	{
		color[0] = min;
		color[1] = max;
		color[2] = posSlope*x-2.0*max+min;
		return;
	}
	else if ( x < 240  )
	{
		color[0] = min;
		color[1] = negSlope*x+4.0*max+min;
		color[2] = max;
		return;
	}
	else if ( x < 300  )
	{
		color[0] = posSlope*x-4.0*max+min;
		color[1] = min;
		color[2] = max;
		return;
	}
	else
	{
		color[0] = max;
		color[1] = min;
		color[2] = negSlope*x+6*max;
		return;
	}
}

/* Mandelbrot Set Image Demonstration
 *
 * This is a simple single-process/single thread implementation
 * that computes a mandelbrot set and produces a corresponding
 * Bitmap image. The program demonstrates the use of a colour
 * gradient
 *
 * This program uses the algorithm outlined in:
 *   "Building Parallel Programs: SMPs, Clusters And Java", Alan Kaminsky
 *
 * This program requires libbmp for all bitmap operations.
 *
 */

/*
 * Function to read in height and width values for the bmp
 * file to be produce. Exits if less than 3 arguments given.
 */
void getParameters(int argc, char** argv, int* height, int* width, size_t* size){
	if (argc < 3 || argv[1] < 0 || argv[2] < 0){
		printf("Usage: <mandelbrot_parallel> <height> <width> \n\n");
		exit(EXIT_SUCCESS);
	}

	*height = atoi(argv[1]);
	*width = atoi(argv[2]);
	*size = *height * *width;
};

/*
 * Structure to hold metadata needed for the production of a series
 * of pixels for Mandelbrot fractal.
 */
typedef struct {
	int width;
	int height;
	float xcenter;
	float ycenter;
	float resolution;
	int iterations;
} Mandelbrot;

/*
 * Kernel functions which returns an array to output of 'iter' values
 * to be used in color function.
 */
/*
	 * 	KERNEL FUNCTION TO POPULATE VALUES
	 */
//col and row values to be worked out from thread position?
//double x = XCENTER + (xoffset + col) / RESOLUTION;
//double y = YCENTER + (yoffset - row) / RESOLUTION;

/*
 * We are going to want a thread to do each of the pixels. Therefore
 * we're going to need x*y threads giving us (x*y)/1024 blocks.
 *
 * If we structure the blocks in a 2d grid, we can just have them
 * pluck out their x,y pixel based on their own x,y location.
 *
 * They can return to x+y...this causes a clash...
 *
 * To do the calculation each thread will need an array of
 * x values and an array of y values.
 *
 * To calculate x we need XCENTER, RESOLUTION, and xoffset.
 * To calculate y we need YCENTER, RESOLUTION, and yoffset.
 *
 * We want to return an iter value corresponding to that pixel that
 * it represents.
 */

__global__ void MandelbrotFractal(float* output, Mandelbrot M)
{

	//get information from 2D block/thread grid
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	//we're only interested in processing threads that fall within
	//the boundaries of the picure
	if (row < M.width && col < M.height){
		int xoffset = -(M.width - 1) /2;
		int yoffset = (M.height -1) / 2;

		//Determine where in the mandelbrot set, the pixel is referencing
		double x = M.xcenter + (xoffset + col) / M.resolution;
		double y = M.ycenter + (yoffset - row) / M.resolution;

		//Mandelbrot stuff
		double a = 0;
		double b = 0;
		double aold = 0;
		double bold = 0;
		double zmagsqr = 0;
		int iter = 0; //import one!

		//Check if the x,y coord are part of the mendelbrot set - refer to the algorithm
		while(iter < M.iterations && zmagsqr <= 4.0){
			++iter;
			a = (aold * aold) - (bold * bold) + x;
			b = 2.0 * aold*bold + y;

			zmagsqr = a*a + b*b;

			aold = a;
			bold = b;
		}
		//output is a 1D array, so we need to index using our row and
		//column number
		output[row * M.width + col] = iter;

	}
}

/*
 * Function to package globals for easier sending to
 * device.
 */
void makeMandel(MandelBrot* M){
	M.iterations = MAX_ITER;
	M.resolution = RESOLUTION;
	M.ycenter = YCENTER;
	M.xcenter = XCENTER;
}

int main(int argc, char **argv)
{
	int height, width;
	size_t size;

	hipError_t error;

	getParameters(argc, argv, &height, &width, &size);

	bmpfile_t *bmp;
	rgb_pixel_t pixel = {0, 0, 0, 0};
	int xoffset = -(width - 1) /2;
	int yoffset = (height -1) / 2;
	bmp = bmp_create(width, height, 32);

	Mandelbrot h_mandel;
	makeMandel(&h_mandel);
	h_mandel.width = width;
	h_mandel.height = height;


	//memory to hold results
	float* h_xy = ( float*) malloc (size);

	//allocate device memory
	float* d_xy;

	error = hipMalloc(&d_xy, size);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_xy returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	Mandelbrot* d_Mandel;

	error = hipMalloc(&d_Mandel, sizeof(Mandelbrot));
	if (error != hipSuccess)
	{
		printf("hipMalloc d_Mandel returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	//copy Mandelbrot metadata to device
	error = hipMemcpy(d_Mandel, h_mandel, sizeof(Mandelbrot), hipMemcpyHostToDevice);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (d_Mandel,h_mandel) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}


	//figure out blocks
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	//figure out threads
	dim3 dimGrid(h_mandel.width / dimBlock.x + 1, h_mandel.height / dimBlock.y + 1);

	//call kernel function
	MandelbrotFractal<<<dimGrid, dimBlock>>>(d_xy, d_Mandel);

	//get data from kernel to device
	error = hipMemcpy(h_xy, d_xy, size, hipMemcpyDeviceToHost);
	if (error != hipSuccess)
	{
		printf("hipMemcpy (h_xy,d_xy) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}


	/* Generate the colour of the pixel from the **iter** value */
	/* You can mess around with the colour settings to use different gradients */
	/* Colour currently maps from royal blue to red */
	/* We're interested in iter */
	int i;
	for (i = 0; i < width*height; i++){
		x_col =  (COLOUR_MAX - (( ((float) iter / ((float) MAX_ITER) * GRADIENT_COLOUR_MAX))));
		GroundColorMix(color, x_col, 1, COLOUR_DEPTH);
		pixel.red = color[0];
		pixel.green = color[1];
		pixel.blue = color[2];

		int row = i/width;
		int col = i % width;

		//adds pixel color to image
		bmp_set_pixel(bmp, col, row, pixel);
	}


	bmp_save(bmp, FILENAME);

	//free all memory used
	bmp_destroy(bmp);

	free(h_xy);
	free(h_mandel);

	hipFree(d_xy);
	hipFree(d_Mandel);

	return 0;
}
